#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "thrust/device_vector.h"

#include "caffe/common_layers.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void kernel_channel_max(const int num, const int channels, const int spatial_dim, const Dtype* data, Dtype* out) {
	for(int index = blockIdx.x * blockDim.x + threadIdx.x; index < num * spatial_dim; index += blockDim.x * gridDim.x){
		int number = index / spatial_dim;
		int spatial = index % spatial_dim;
		Dtype result = -FLT_MAX;
		for (int chanel = 0; chanel < channels; ++chanel) {
		  result = max(data[(number * channels + chanel) * spatial_dim + spatial], result);
		}
		out[index] = result;
	}
}

template <typename Dtype>
__global__ void kernel_channel_subtract(const int count, const int num, const int channels, const int spatial_dim, const Dtype* channel_max, Dtype* data) {
	for(int index = blockIdx.x * blockDim.x + threadIdx.x; index < count; index += blockDim.x * gridDim.x){
		int number = index / channels / spatial_dim;
		int spatial = index % spatial_dim;
		data[index] = data[index] - channel_max[number * spatial_dim + spatial];
  }
}

template <typename Dtype>
__global__ void kernel_exp(const int count, const Dtype* data, Dtype* out) {
	for(int index = blockIdx.x * blockDim.x + threadIdx.x; index < count; index += blockDim.x * gridDim.x){
		out[index] = exp(data[index]);
  }
}

template <typename Dtype>
__global__ void kernel_channel_sum(const int num, const int channels, const int spatial_dim, const Dtype* data, Dtype* channel_sum) {
	for(int index = blockIdx.x * blockDim.x + threadIdx.x; index < num * spatial_dim; index += blockDim.x * gridDim.x){
		int number = index / spatial_dim;
		int spatial = index % spatial_dim;
		Dtype sum = 0;
		for (int chanel = 0; chanel < channels; ++chanel) {
		  sum += data[(number * channels + chanel) * spatial_dim + spatial];
		}
		channel_sum[index] = sum;
  }
}

template <typename Dtype>
__global__ void kernel_channel_div(const int count, const int num, const int channels, const int spatial_dim, const Dtype* channel_sum, Dtype* data) {
	for(int index = blockIdx.x * blockDim.x + threadIdx.x; index < count; index += blockDim.x * gridDim.x){
		int number = index / channels / spatial_dim;
		int spatial = index % spatial_dim;
		data[index] = data[index] / channel_sum[number * spatial_dim + spatial];
  }
}

template <typename Dtype>
__global__ void kernel_channel_dot(const int num, const int channels, const int spatial_dim, const Dtype* data_1, const Dtype* data_2,Dtype* channel_dot) {
	for(int index = blockIdx.x * blockDim.x + threadIdx.x; index < num * spatial_dim; index += blockDim.x * gridDim.x){
		int number = index / spatial_dim;
		int spatial = index % spatial_dim;
		int idx;
		Dtype dot = 0;
		for (int chanel = 0; chanel < channels; ++chanel) {
			idx = (number * channels + chanel) * spatial_dim + spatial;
			dot += data_1[idx] * data_2[idx];
		}
		channel_dot[index] = dot;
	}
}

template <typename Dtype>
void SoftmaxLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  Dtype* scale_data = scale_.mutable_gpu_data();
  int count = bottom[0]->count();
  int channels = top[0]->shape(softmax_axis_);
  caffe_copy(count, bottom_data, top_data);
  // We need to subtract the max to avoid numerical issues, compute the exp,
  // and then normalize.
  // compute max
  // NOLINT_NEXT_LINE(whitespace/operators)
  kernel_channel_max<Dtype><<<CAFFE_GET_BLOCKS(outer_num_ * inner_num_),
      CAFFE_CUDA_NUM_THREADS>>>(outer_num_, channels, inner_num_, top_data,
      scale_data);
  // subtract
  // NOLINT_NEXT_LINE(whitespace/operators)
  kernel_channel_subtract<Dtype><<<CAFFE_GET_BLOCKS(count),
      CAFFE_CUDA_NUM_THREADS>>>(count, outer_num_, channels, inner_num_,
      scale_data, top_data);
  // exponentiate
  // NOLINT_NEXT_LINE(whitespace/operators)
  kernel_exp<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, top_data, top_data);
  // sum after exp
  // NOLINT_NEXT_LINE(whitespace/operators)
  kernel_channel_sum<Dtype><<<CAFFE_GET_BLOCKS(outer_num_ * inner_num_),
      CAFFE_CUDA_NUM_THREADS>>>(outer_num_, channels, inner_num_, top_data,
      scale_data);
  // divide
  // NOLINT_NEXT_LINE(whitespace/operators)
  kernel_channel_div<Dtype><<<CAFFE_GET_BLOCKS(count),
      CAFFE_CUDA_NUM_THREADS>>>(count, outer_num_, channels, inner_num_,
      scale_data, top_data);
}

template <typename Dtype>
void SoftmaxLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const Dtype* top_diff = top[0]->gpu_diff();
  const Dtype* top_data = top[0]->gpu_data();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  Dtype* scale_data = scale_.mutable_gpu_data();
  int count = top[0]->count();
  int channels = top[0]->shape(softmax_axis_);
  caffe_copy(count, top_diff, bottom_diff);
  // Compute inner1d(top_diff, top_data) and subtract them from the bottom diff.
  // NOLINT_NEXT_LINE(whitespace/operators)
  kernel_channel_dot<Dtype><<<CAFFE_GET_BLOCKS(outer_num_ * inner_num_),
      CAFFE_CUDA_NUM_THREADS>>>(outer_num_, channels, inner_num_,
      top_diff, top_data, scale_data);
  // NOLINT_NEXT_LINE(whitespace/operators)
  kernel_channel_subtract<Dtype><<<CAFFE_GET_BLOCKS(count),
      CAFFE_CUDA_NUM_THREADS>>>(count, outer_num_, channels, inner_num_,
      scale_data, bottom_diff);
  // elementwise multiplication
  caffe_gpu_mul<Dtype>(top[0]->count(), bottom_diff, top_data, bottom_diff);
}

INSTANTIATE_LAYER_GPU_FUNCS(SoftmaxLayer);


}  // namespace caffe
